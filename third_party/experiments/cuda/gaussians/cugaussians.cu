#include "hip/hip_runtime.h"
#ifndef _DIST_KERNEL_CU_
#define _DIST_KERNEL_CU_

#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>

#define EIGEN_DEFAULT_DENSE_INDEX_TYPE int
#include <Eigen/Dense>

__device__ uint rgbaFloatToInt(float4 rgba) {
  rgba.x = __saturatef(rgba.x);  // clamp to [0.0, 1.0]
  rgba.y = __saturatef(rgba.y);
  rgba.z = __saturatef(rgba.z);
  rgba.w = __saturatef(rgba.w);
  return (uint(rgba.w * 255) << 24) | (uint(rgba.z * 255) << 16) | (uint(rgba.y * 255) << 8) | uint(rgba.x * 255);
}

__global__ void d_render(uint *           d_output,
                         uint             imageW,
                         uint             imageH,
                         float            scale,
                         float2           view_center,
                         Eigen::Vector2f *means,
                         Eigen::Matrix2f *information_matrices,
                         int              N,
                         float            normalization,
                         float            tstep) {
  uint x = blockIdx.x * blockDim.x + threadIdx.x;
  uint y = blockIdx.y * blockDim.y + threadIdx.y;

  if ((x >= imageW) || (y >= imageH)) {
    return;
  }

  // These int-float cast operations aren't cheap
  const float unscaled_u = ((x / (float)imageW) * 2.0f) - 1.0f;
  const float unscaled_v = ((y / (float)imageH) * 2.0f) - 1.0f;

  const float u = scale * (unscaled_u + view_center.x);
  const float v = scale * (unscaled_v + view_center.y);

  Eigen::Vector2f vv    = Eigen::Vector2f(u, v);
  float           value = 0.0f;
  for (int k = 0; k < N; ++k) {
    const Eigen::Vector2f eigv        = (vv - means[k]);
    const float           mahalanobis = (eigv.transpose() * information_matrices[k] * eigv)(0);
    const float           probability = __expf(-0.5f * mahalanobis);
    value += probability;
  }
  const float normalized_value = value * normalization;
  float4      color =
      make_float4(tstep * normalized_value, normalized_value * 0.5f, tstep * normalized_value * normalized_value, 1.0f);

  // This could be optimized
  d_output[y * imageW + x] = rgbaFloatToInt(color);
}

void render_kernel(dim3             gridSize,
                   dim3             blockSize,
                   uint *           d_output,
                   uint             imageW,
                   uint             imageH,
                   float            scale,
                   float2           view_center,
                   Eigen::Vector2f *means,
                   Eigen::Matrix2f *information_matrices,
                   int              N,
                   float            normalization,
                   float            tstep) {
  d_render<<<gridSize, blockSize>>>(
      d_output, imageW, imageH, scale, view_center, means, information_matrices, N, normalization, tstep);
}

#endif  // #ifndef _DIST_KERNEL_CU_
