#include "hip/hip_runtime.h"
#ifndef _DIST_KERNEL_CU_
#define _DIST_KERNEL_CU_

#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>

#define EIGEN_DEFAULT_DENSE_INDEX_TYPE int
#include <Eigen/Dense>

__device__ uint pack_frgba(float4 rgba) {
  rgba.x = __saturatef(rgba.x);  // clamp to [0.0, 1.0]
  rgba.y = __saturatef(rgba.y);
  rgba.z = __saturatef(rgba.z);
  rgba.w = __saturatef(rgba.w);
  return (uint(rgba.w * 255) << 24) | (uint(rgba.z * 255) << 16) |
         (uint(rgba.y * 255) << 8) | uint(rgba.x * 255);
}

__device__ float sd_sphere(const Eigen::Vector3f &center,
                           const float radius,
                           const Eigen::Vector3f &ray_dir,
                           const Eigen::Vector3f &ray_origin) {
  float min_dist = 1000.0f;
  for (float t = 0.0f; t < 10.0f; t += 0.1f) {
    const Eigen::Vector3f error = ((t * ray_dir) + ray_origin) - center;
    const float sd_sphere = error.norm() - radius;
    min_dist = fminf(min_dist, sd_sphere);
  }
  return min_dist;
}

__global__ void d_render(uint *const d_output,
                         const uint imageW,
                         const uint imageH,
                         const Eigen::Vector3f *const centers,
                         const float *const radii,
                         const int N,
                         const float normalization,
                         const float t) {
  const uint x = blockIdx.x * blockDim.x + threadIdx.x;
  const uint y = blockIdx.y * blockDim.y + threadIdx.y;

  if ((x >= imageW) || (y >= imageH)) {
    return;
  }

  // These int-float cast operations aren't cheap
  const float unscaled_u = ((x / (float)imageW) * 2.0f) - 1.0f;
  const float unscaled_v = ((y / (float)imageH) * 2.0f) - 1.0f;

  const float2 view_center = make_float2(0.0f, 0.0f);
  const float scale = 1.0;

  const float u = scale * (unscaled_u + view_center.x);
  const float v = scale * (unscaled_v + view_center.y);

  float max_dist = 10000.0f;
  const Eigen::Vector3f pixel_position(u, v, 0.0f);
  const Eigen::Vector3f pixel_ray = Eigen::Vector3f(u, v, 1.0f).normalized();

  const Eigen::AngleAxisf rot(t, Eigen::Vector3f::UnitZ());

  for (uint k = 0u; k < N; ++k) {
    const float distance =
        sd_sphere(rot * centers[k], radii[k], pixel_ray, pixel_position);
    max_dist = fminf(max_dist, distance);
    // max_dist = fmaxf(max_dist, distance);
  }

  float4 color;
  // color.x = max_dist * normalization;
  color.x = max_dist < 0.0f ? 1.0f : 0.0f;

  color.y = 0.0f;
  color.z = 0.0f;
  color.w = 1.0f;

  // This could be optimized
  d_output[y * imageW + x] = pack_frgba(color);
}

void render_kernel(dim3 gridSize,
                   dim3 blockSize,
                   uint *const d_output,
                   const uint imageW,
                   const uint imageH,
                   const Eigen::Vector3f *const centers,
                   const float *const radii,
                   const int N,
                   const float normalization,
                   const float t) {
  d_render<<<gridSize, blockSize>>>(
      d_output, imageW, imageH, centers, radii, N, normalization, t);
}

#endif  // #ifndef _DIST_KERNEL_CU_
