#include "hip/hip_runtime.h"
#ifndef _DIST_KERNEL_CU_
#define _DIST_KERNEL_CU_

#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>

__device__ uint rgbaFloatToInt(float4 rgba) {
  rgba.x = __saturatef(rgba.x);  // clamp to [0.0, 1.0]
  rgba.y = __saturatef(rgba.y);
  rgba.z = __saturatef(rgba.z);
  rgba.w = __saturatef(rgba.w);
  return (uint(rgba.w * 255) << 24) | (uint(rgba.z * 255) << 16) |
         (uint(rgba.y * 255) << 8) | uint(rgba.x * 255);
}

__global__ void d_render(uint *d_output, uint imageW, uint imageH, float scale,
                         float2 view_center) {
  uint x = blockIdx.x * blockDim.x + threadIdx.x;
  uint y = blockIdx.y * blockDim.y + threadIdx.y;

  if ((x >= imageW) || (y >= imageH)) {
    return;
  }

  // These int-float cast operations aren't cheap
  const float unscaled_u = ((x / (float)imageW) * 2.0f) - 1.0f;
  const float unscaled_v = ((y / (float)imageH) * 2.0f) - 1.0f;

  const float u = scale * (unscaled_u + view_center.x);
  const float v = scale * (unscaled_v + view_center.y);

  float4 sum = make_float4(0.0f);
  const float distance = hypot(u, v);
  sum.x = distance;
  sum.y = distance / sqrt(distance) * v;
  sum.z = distance * distance;
  sum.w = 1.0;

  // This could be optimized
  d_output[y * imageW + x] = rgbaFloatToInt(sum);
}

extern "C" void render_kernel(dim3 gridSize, dim3 blockSize, uint *d_output,
                              uint imageW, uint imageH, float scale,
                              float2 view_center) {
  d_render<<<gridSize, blockSize>>>(d_output, imageW, imageH, scale,
                                    view_center);
}

#endif  // #ifndef _DIST_KERNEL_CU_
